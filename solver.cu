#include "hip/hip_runtime.h"
#include <stddef.h>

#include "solver.h"
#include "indices.cuh"

#define BLOCK_SIZE 128


#define IX(x,y) (rb_idx((x),(y),(n+2)))
#define SWAP(x0,x) {float * tmp=x0;x0=x;x=tmp;}

typedef enum { NONE = 0, VERTICAL = 1, HORIZONTAL = 2 } boundary;
typedef enum { RED, BLACK } grid_color;

__global__ void kernel_add_source(float * x, const float * s, float dt, unsigned int size)
{
    size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= size) return;
    x[gid] += dt * s[gid];
}

static void add_source(unsigned int n, float * x, const float * s, float dt)
{
    unsigned int size = (n + 2) * (n + 2);
    kernel_add_source<<<BLOCK_SIZE, (size+127)/BLOCK_SIZE>>>(x, s, dt, size);
    hipDeviceSynchronize();
    /*for (unsigned int i = 0; i < size; i++) {
        x[i] += dt * s[i];
    }*/
}

__global__ void kernel_set_bnd(unsigned int n, boundary b, float * x)
{
    size_t gid = blockIdx.x * blockDim.x + threadIdx.x + 1;

    if(gid > n) return;
    
    if(gid == 0){
        x[IX(0, 0)]         = 0.5f * (x[IX(1, 0)]     + x[IX(0, 1)]);
        x[IX(0, n + 1)]     = 0.5f * (x[IX(1, n + 1)] + x[IX(0, n)]);
        x[IX(n + 1, 0)]     = 0.5f * (x[IX(n, 0)]     + x[IX(n + 1, 1)]);
        x[IX(n + 1, n + 1)] = 0.5f * (x[IX(n, n + 1)] + x[IX(n + 1, n)]);
    }else{
        x[IX(0, gid)]     = b == VERTICAL ? -x[IX(1, gid)] : x[IX(1, gid)];
        x[IX(n + 1, gid)] = b == VERTICAL ? -x[IX(n, gid)] : x[IX(n, gid)];
        x[IX(gid, 0)]     = b == HORIZONTAL ? -x[IX(gid, 1)] : x[IX(gid, 1)];
        x[IX(gid, n + 1)] = b == HORIZONTAL ? -x[IX(gid, n)] : x[IX(gid, n)];
    }
}

static void set_bnd(unsigned int n, boundary b, float * x)
{
    
    kernel_set_bnd<<<BLOCK_SIZE, ((n+1)+(BLOCK_SIZE-1))/BLOCK_SIZE>>>(n, b, x);
    hipDeviceSynchronize();
    
    // for (unsigned int i = 1; i <= n; i++) {
        //     x[IX(0, i)]     = b == VERTICAL ? -x[IX(1, i)] : x[IX(1, i)];
        //     x[IX(n + 1, i)] = b == VERTICAL ? -x[IX(n, i)] : x[IX(n, i)];
        //     x[IX(i, 0)]     = b == HORIZONTAL ? -x[IX(i, 1)] : x[IX(i, 1)];
        //     x[IX(i, n + 1)] = b == HORIZONTAL ? -x[IX(i, n)] : x[IX(i, n)];
        // }
        // x[IX(0, 0)]         = 0.5f * (x[IX(1, 0)]     + x[IX(0, 1)]);
        // x[IX(0, n + 1)]     = 0.5f * (x[IX(1, n + 1)] + x[IX(0, n)]);
        // x[IX(n + 1, 0)]     = 0.5f * (x[IX(n, 0)]     + x[IX(n + 1, 1)]);
        // x[IX(n + 1, n + 1)] = 0.5f * (x[IX(n, n + 1)] + x[IX(n + 1, n)]);
}

__global__ void kernel_lin_solve_rb_step(grid_color color,
                                         float a,
                                         float c,
                                         const float * same0,
                                         const float * neigh,
                                         float * same){
    size_t bid = blockIdx.x;
    size_t tid = threadIdx.x;
    unsigned int width = blockDim.x + 1;

    int shift = color == RED ? 1 : -1;
    unsigned int start = color == RED ? 0 : 1;
    
    unsigned int y = bid + 1;
    if(y % 2 == 0) {
        shift = -shift;
        start = 1 - start;
    }

    /*__shared__ float shared_abv_rgt[1024];
    if(tid==0){
        for(int i=0;i<width;i++){
            shared_abv_rgt[i] = neigh[y*width + i];
        }
    }
    __syncthreads();*/

    float* row_same = same + y*width;
    const float* row_same0 = same0 + y*width;
    const float* lft = neigh + y*width - width;
    //const float* abv = shared_abv_rgt;
    //const float* rgt = shared_abv_rgt + shift;
    const float* abv = neigh + y*width;
    const float* rgt = neigh + y*width + shift;
    const float* blw = neigh + y*width + width;

    unsigned int x = tid + start;
    row_same[x] = (row_same0[x] + a * (lft[x] +
                                        abv[x] +
                                        rgt[x] +
                                        blw[x])) / c;
}

static void lin_solve_rb_step(grid_color color,
                              unsigned int n,
                              float a,
                              float c,
                              const float * same0,
                              const float * neigh,
                              float * same)
{
    /*int shift = color == RED ? 1 : -1;
    unsigned int start = color == RED ? 0 : 1;*/

    /*unsigned int width = (n + 2) / 2;
 
    for (unsigned int y = 1; y <= n; ++y, shift = -shift, start = 1 - start) { 
        float* row_same = same + y*width;
        const float* row_same0 = same0 + y*width;
        const float* lft = neigh + y*width - width;
        const float* abv = neigh + y*width;
        const float* rgt = neigh + y*width + shift;
        const float* blw = neigh + y*width + width;

        for (unsigned int x = start; x < width - (1 - start); ++x) {
            row_same[x] = (row_same0[x] + a * (lft[x] +
                                               abv[x] +
                                               rgt[x] +
                                               blw[x])) / c;
        }
    }*/
    unsigned int width = (n + 2) / 2;
    kernel_lin_solve_rb_step<<<n, (width - 1)>>>(color, a, c, same0, neigh, same);
    hipDeviceSynchronize();
}

static void lin_solve(unsigned int n, boundary b,
                      float * x,
                      const float * x0,
                      float a, float c)
{
    unsigned int color_size = (n + 2) * ((n + 2) / 2);
    const float * red0 = x0;
    const float * blk0 = x0 + color_size;
    float * red = x;
    float * blk = x + color_size;

    for (unsigned int k = 0; k < 20; ++k) {
        lin_solve_rb_step(RED, n, a, c, red0, blk, red);
        lin_solve_rb_step(BLACK, n, a, c, blk0, red, blk);
        set_bnd(n, b, x);
    }

}

static void diffuse(unsigned int n, boundary b, float * x, const float * x0, float diff, float dt)
{
    float a = dt * diff * n * n;
    lin_solve(n, b, x, x0, a, 1 + 4 * a);
}

__global__ void kernel_advect(float dt0, float * d, const float* u, const float * v, const float * d0)
{
    size_t n = blockDim.x;
    size_t i = blockIdx.x + 1;
    size_t j = threadIdx.x + 1; 
    
    float x = i - dt0 * u[IX(i, j)];
    float y = j - dt0 * v[IX(i, j)];
    if (x < 0.5f) {
        x = 0.5f;
    } else if (x > n + 0.5f) {
        x = n + 0.5f;
    }
    float i0 = (int) x;
    float i1 = i0 + 1;
    if (y < 0.5f) {
        y = 0.5f;
    } else if (y > n + 0.5f) {
        y = n + 0.5f;
    }
    float j0 = (int) y;
    float j1 = j0 + 1;
    float s1 = x - i0;
    float s0 = 1 - s1;
    float t1 = y - j0;
    float t0 = 1 - t1;
    d[IX(i, j)] = s0 * (t0 * d0[IX(i0, j0)] + t1 * d0[IX(i0, j1)]) +
                        s1 * (t0 * d0[IX(i1, j0)] + t1 * d0[IX(i1, j1)]);
}

static void advect(unsigned int n, boundary b, float * d, const float * d0, const float * u, const float * v, float dt)
{
    // int i0, i1, j0, j1;
    // float x, y, s0, t0, s1, t1;

    float dt0 = dt * n;
    kernel_advect<<<n, n>>>(dt0,d,u,v,d0);
    hipDeviceSynchronize();
    
    // for (unsigned int i = 1; i <= n; i++) {
    //     for (unsigned int j = 1; j <= n; j++) {
    //         x = i - dt0 * u[IX(i, j)];
    //         y = j - dt0 * v[IX(i, j)];
    //         if (x < 0.5f) {
    //             x = 0.5f;
    //         } else if (x > n + 0.5f) {
    //             x = n + 0.5f;
    //         }
    //         i0 = (int) x;
    //         i1 = i0 + 1;
    //         if (y < 0.5f) {
    //             y = 0.5f;
    //         } else if (y > n + 0.5f) {
    //             y = n + 0.5f;
    //         }
    //         j0 = (int) y;
    //         j1 = j0 + 1;
    //         s1 = x - i0;
    //         s0 = 1 - s1;
    //         t1 = y - j0;
    //         t0 = 1 - t1;
    //         d[IX(i, j)] = s0 * (t0 * d0[IX(i0, j0)] + t1 * d0[IX(i0, j1)]) +
    //                       s1 * (t0 * d0[IX(i1, j0)] + t1 * d0[IX(i1, j1)]);
    //     }
    // }
    set_bnd(n, b, d);
}

__global__ void kernel_uno_project(float * u, float * v, float * p, float * div)
{
    size_t n = blockDim.x;
    size_t i = blockIdx.x + 1;
    size_t j = threadIdx.x + 1; 

    div[IX(i, j)] = -0.5f * (u[IX(i + 1, j)] - u[IX(i - 1, j)] +
                             v[IX(i, j + 1)] - v[IX(i, j - 1)]) / n;
    p[IX(i, j)] = 0;
}

__global__ void kernel_dos_project(float * u, float * v, float * p)
{
    size_t n = blockDim.x;
    size_t i = blockIdx.x + 1;
    size_t j = threadIdx.x + 1; 

    u[IX(i, j)] -= 0.5f * n * (p[IX(i + 1, j)] - p[IX(i - 1, j)]);
    v[IX(i, j)] -= 0.5f * n * (p[IX(i, j + 1)] - p[IX(i, j - 1)]);
}


static void project(unsigned int n, float *u, float *v, float *p, float *div)
{
    kernel_uno_project<<<n, n>>>(u, v, p, div);
    hipDeviceSynchronize();
    // for (unsigned int i = 1; i <= n; i++) {
    //     for (unsigned int j = 1; j <= n; j++) {
    //         div[IX(i, j)] = -0.5f * (u[IX(i + 1, j)] - u[IX(i - 1, j)] +
    //                                  v[IX(i, j + 1)] - v[IX(i, j - 1)]) / n;
    //         p[IX(i, j)] = 0;
    //     }
    // }
    set_bnd(n, NONE, div);
    set_bnd(n, NONE, p);

    lin_solve(n, NONE, p, div, 1, 4);

    kernel_dos_project<<<n, n>>>(u, v, p);
    hipDeviceSynchronize();
    // for (unsigned int i = 1; i <= n; i++) {
    //     for (unsigned int j = 1; j <= n; j++) {
    //         u[IX(i, j)] -= 0.5f * n * (p[IX(i + 1, j)] - p[IX(i - 1, j)]);
    //         v[IX(i, j)] -= 0.5f * n * (p[IX(i, j + 1)] - p[IX(i, j - 1)]);
    //     }
    // }
    set_bnd(n, VERTICAL, u);
    set_bnd(n, HORIZONTAL, v);
}

void dens_step(unsigned int n, float *x, float *x0, float *u, float *v, float diff, float dt)
{
    add_source(n, x, x0, dt);
    SWAP(x0, x);
    diffuse(n, NONE, x, x0, diff, dt);
    SWAP(x0, x);
    advect(n, NONE, x, x0, u, v, dt);
}

void vel_step(unsigned int n, float *u, float *v, float *u0, float *v0, float visc, float dt)
{
    add_source(n, u, u0, dt);
    add_source(n, v, v0, dt);
    SWAP(u0, u);
    diffuse(n, VERTICAL, u, u0, visc, dt);
    SWAP(v0, v);
    diffuse(n, HORIZONTAL, v, v0, visc, dt);
    project(n, u, v, u0, v0);
    SWAP(u0, u);
    SWAP(v0, v);
    advect(n, VERTICAL, u, u0, u0, v0, dt);
    advect(n, HORIZONTAL, v, v0, u0, v0, dt);
    project(n, u, v, u0, v0);
}
